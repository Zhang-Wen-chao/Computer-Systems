
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("Hello from GPU\n");
}

int main() {
    printf("Hello from CPU\n");
    
    kernel<<<1,1>>>();
    
    hipDeviceReset(); // if no this line, it can not output hello world from gpu
    
    return 0;
}
